#include "hip/hip_runtime.h"
#include "gpuplugin.h"

// GPU functions

void checkGPUInfo() {
    int deviceCount, device;
    struct hipDeviceProp_t properties;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    if (cudaResultCode != hipSuccess)
        deviceCount = 0;
    /* machines with no GPUs can still report one emulation device */
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties, device);
        if (properties.major != 9999) /* 9999 means emulation only */
            if (device == 0) {
                printf("multiProcessorCount %d\n",
                       properties.multiProcessorCount);
                printf("maxThreadsPerMultiProcessor %d\n",
                       properties.maxThreadsPerMultiProcessor);
            }
    }
}

void particlesToUM(vector<Particle*> particles, long* idUM, double* massUM,
                   double* posUM, double* velUM, double* accUM) {
    long N = particles.size();
    for (long i = 0; i < N; i++) {
        idUM[i] = particles[i]->id;
        massUM[i] = particles[i]->mass;
        for (int j = 0; j < 3; j++) {
            posUM[i * 3 + j] = particles[i]->pos[j];
            velUM[i * 3 + j] = particles[i]->vel[j];
            accUM[i * 3 + j] = particles[i]->acc[j];
        }
    }
}

void UMToParticles(vector<Particle*> particles, long* idUM, double* massUM,
                   double* posUM, double* velUM, double* accUM) {
    long N = particles.size();
    for (long i = 0; i < N; i++) {
        particles[i]->id = idUM[i];
        particles[i]->mass = massUM[i];
        for (int j = 0; j < 3; j++) {
            particles[i]->pos[j] = posUM[i * 3 + j];
            particles[i]->vel[j] = velUM[i * 3 + j];
            particles[i]->acc[j] = accUM[i * 3 + j];
        }
    }
}

void checkUM(vector<Particle*> particles, long* idUM, double* massUM,
             double* posUM, double* velUM, double* accUM) {
    long N = particles.size();
    for (long i = 0; i < N; i++) {
        assert(particles[i]->id == idUM[i] && "IDs not match!");
        assert(particles[i]->mass == massUM[i] && "Masses not match!");
        for (int j = 0; j < 3; j++) {
            assert(particles[i]->pos[j] == posUM[i * 3 + j] &&
                   "Positions not match!");
            assert(particles[i]->vel[j] == velUM[i * 3 + j] &&
                   "Velocities not match!");
            assert(particles[i]->acc[j] == accUM[i * 3 + j] &&
                   "Acclerations not match!");
        }
    }
}

__global__ void initializeAcc(long N, double* accUM) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (long i = 0; i < N; i += stride) {
        if (tid + i < N) {
            for (int j = 0; j < 3; j++) {
                accUM[(tid + i) * 3 + j] = 0.0;
            }
        }
    }
}

__global__ void move(long N, double* posUM, double* velUM, double* accUM) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (long i = 0; i < N; i += stride) {
        if (tid + i < N) {
            for (int j = 0; j < 3; j++) {
                atomicAdd(&velUM[(tid + i) * 3 + j],
                          accUM[(tid + i) * 3 + j] * TIMESTEP);
                atomicAdd(&posUM[(tid + i) * 3 + j],
                          velUM[(tid + i) * 3 + j] * TIMESTEP);
            }
        }
    }
}

// ----
__global__ void updateGravityBruteMethod(long N, double* massUM, double* posUM,
                                         double* velUM, double* accUM) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (long i = 0; i < N; i += stride) {
        if (tid + i < N) {
            for (long k = tid + i + 1; k < N; k++) {
                // particles pair
                double rVector[3];
                double rNorm = 0.0;
                for (int j = 0; j < 3; j++) {
                    rVector[j] = posUM[(tid + i) * 3 + j] - posUM[k * 3 + j];
                    rNorm += pow(rVector[j], 2.0);
                }
                rNorm = sqrt(rNorm);
                assert(rNorm != 0.0 || SOFTEN_FACTOR != 0.0);
                for (int j = 0; j < 3; j++) {
                    atomicAdd(
                        &accUM[(tid + i) * 3 + j],
                        -CONST_G * massUM[k] * rVector[j] /
                            pow((pow(rNorm, 2.0) + pow(SOFTEN_FACTOR, 2.0)),
                                1.5));
                    atomicAdd(
                        &accUM[k * 3 + j],
                        CONST_G * massUM[tid + i] * rVector[j] /
                            pow((pow(rNorm, 2.0) + pow(SOFTEN_FACTOR, 2.0)),
                                1.5));
                }
            }
        }
    }
    return;
}

void updateBruteMethodGPU(long N, double* massUM, double* posUM, double* velUM,
                          double* accUM) {

    initializeAcc<<<GRIDDIM, BLOCKDIM>>>(N, accUM);
    hipDeviceSynchronize();

    updateGravityBruteMethod<<<GRIDDIM, BLOCKDIM>>>(N, massUM, posUM, velUM,
                                                    accUM);
    hipDeviceSynchronize();
    move<<<GRIDDIM, BLOCKDIM>>>(N, posUM, velUM, accUM);
    hipDeviceSynchronize();
    return;
}

// ================= Example =============== //

// __global__ void test(double posGPU[100000][3]) {
//     int id = blockIdx.x * blockDim.x + threadIdx.x;
//     // printf("%lf\n", posGPU[400][2]);
// }

// int main() {
//     // CUDA UnifinedMemory Notes:
//     // 1. new -> hipMallocManaged() -> hipDeviceSynchronize() ->
//     hipFree()
//     ->
//     // free()

//     vector<Particle*> particles = readParticles("particles_N1000.txt");
//     // const long N = particles.size();
//     const long N = 1000000;

//     auto* posGPU = new double[N][3];
//     hipMallocManaged((void**)&posGPU, sizeof(double) * N * 3);
//     for (int i = 0; i < N; i++) {
//         for (int j = 0; j < 3; j++) {
//             posGPU[i][j] = i * j;
//         }
//     }

//     test<<<GRIDDIM, BLOCKDIM>>>(posGPU);
//     hipDeviceSynchronize();
//     return 0;
// }
// ================================================= //