#include "hip/hip_runtime.h"
#include "fileIO.h"
#include "gpuplugin.h"
#include "particle.h"
#include "tree.h"
#include "updater.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/types.h>
#include <time.h>
#include <vector>
using namespace std;

// notice
// 1. all particles must be declared as { Particle* p = new Particle(id, mass) }
// 2. or use static (assure it will not be deleted)

char* createOutputFolder() {
    struct timeval tp;
    static char outputFolder[50];
    gettimeofday(&tp, NULL);
    sprintf(outputFolder, "output-%ld", tp.tv_sec);
    mkdir(outputFolder, 0755);
    return outputFolder;
}

int main() {
    clock_t clkStart, clkFinish;

    // CUDA UnifinedMemory Notes:
    // 1. new -> hipMallocManaged() -> hipDeviceSynchronize() -> hipFree() ->
    // free()

    // frame counter
    long frame = 0;
    // create output folder to store result
    char* outputFolder = createOutputFolder();

    vector<Particle*> particles =
        readParticles("inputs/P10000New/P-N10000.txt");

    clkStart = clock();
    for (int i = 0; i < 300; i++) {
        Node* root = new Node(particles);
        // clkStart = clock();
        buildTree(root);
        // clkFinish = clock();
        // cout << "Runtime: " << (clkFinish - clkStart) * 1e-6 << " s" << endl;
        saveData(root, outputFolder, frame, true, false, true);
        updateTreeMethod(root);
        // updateBruteMethod(root);
        deleteTree(root);
        frame++;
    }
    clkFinish = clock();

    // vector<Particle*> particles;
    // for (int i = 0; i < 1000; i++) {
    //     Particle* p = new Particle(i, 1.0);
    //     p->setRandomPosition(-100, 100);
    //     particles.push_back(p);
    // }
    // printf("Generate N particles\n");

    // const long N = particles.size();

    // // CUDA Unified Memory (1D linearlized array)
    // long* idUM = new long[N];
    // double* massUM = new double[N];
    // double* posUM = new double[N * 3];
    // double* velUM = new double[N * 3];
    // double* accUM = new double[N * 3];
    // hipMallocManaged((void**)&idUM, sizeof(long) * N);
    // hipMallocManaged((void**)&massUM, sizeof(double) * N);
    // hipMallocManaged((void**)&posUM, sizeof(double) * N * 3);
    // hipMallocManaged((void**)&velUM, sizeof(double) * N * 3);
    // hipMallocManaged((void**)&accUM, sizeof(double) * N * 3);
    // particlesToUM(particles, idUM, massUM, posUM, velUM, accUM);
    // checkUM(particles, idUM, massUM, posUM, velUM, accUM);

    // clkStart = clock();
    // for (int i = 0; i < 10; i++) {

    //     // Node root(particles);
    //     // buildTree(&root);
    //     // updateTreeMethod(&root);

    //     updateBruteMethodGPU(N, massUM, posUM, velUM, accUM);

    //     cout << i << endl;
    // }
    // clkFinish = clock();

    // UMToParticles(particles, idUM, massUM, posUM, velUM, accUM);
    // particles[10]->info();

    cout << "Runtime: " << (clkFinish - clkStart) * 1e-6 << " s" << endl;
    return 0;
}
